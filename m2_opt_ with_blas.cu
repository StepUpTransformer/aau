#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <sys/time.h>

// CUDA Kernel for matrix-vector multiplication with bias
// Grid size: (5000,1,1), Block size: (32,4,1)
__global__ void mm_cuda(const float* __restrict__ mat, const float* __restrict__ vec, float* __restrict__ output, int rows, int cols) {
    extern __shared__ float shared_vec[];
    int row = (blockIdx.x * blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    if (threadIdx.x < cols) {
        shared_vec[threadIdx.x] = vec[threadIdx.x];
    }
    __syncthreads();

    if (row < rows) {
        float mul = 0.0f;
        for (int col = 0; col < cols; ++col) {
            mul += mat[row * cols + col] * shared_vec[col];
        }
        output[row] = mul;
    }
}

// CUDA Kernel for ReLU activation
// Grid size: (79,1,1), Block size: (128,1,1)
__global__ void ReLU_cuda(float* __restrict__ data, const float* __restrict__ bias, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float val = data[idx] + bias[idx];
        data[idx] = fmaxf(val, 0.0f);
    }
}

// CUDA Kernel for matrix-vector addition with bias
// Grid size: (25,1,4), Block size: (34,4,1)
__global__ void addmm_cuda(const float* __restrict__ mat, const float* __restrict__ vec, const float* __restrict__ bias, float* __restrict__ output, int rows, int cols) {
    extern __shared__ float shared_data[];
    float* shared_vec = shared_data;
    float* shared_bias = shared_vec + blockDim.x;

    int tid = threadIdx.x;
    if (tid < cols) {
        shared_vec[tid] = vec[tid];
    }
    if (tid < rows) {
        shared_bias[tid] = bias[tid];
    }
    __syncthreads();

    int row = (blockIdx.x * blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    row += blockIdx.z * (gridDim.x * blockDim.x * blockDim.y);

    if (row < rows) {
        float sum = shared_bias[row];
        for (int col = 0; col < cols; ++col) {
            sum += mat[row * cols + col] * shared_vec[col];
        }
        output[row] = sum;
    }
}

// CUDA Kernel for Softmax computation
// Grid size: (1,1,1), Block size: (64,1,1)
__global__ void softmax_1_cuda(float* __restrict__ output, float* __restrict__ softmax_out, int output_size) {
    int idx = threadIdx.x;

    // Compute maximum using warp reduction
    float max_val = -FLT_MAX;
    if (idx < output_size) {
        max_val = output[idx];
    }
    for (int offset = 32 / 2; offset > 0; offset /= 2) {
        max_val = fmaxf(max_val, __shfl_down_sync(0xffffffff, max_val, offset));
    }

    __shared__ float shared_max;
    if (idx == 0) shared_max = max_val;
    __syncthreads();

    // Compute sum of exponentials
    float sum_exp = 0.0f;
    if (idx < output_size) {
        sum_exp = expf(output[idx] - shared_max);
    }
    for (int offset = 32 / 2; offset > 0; offset /= 2) {
        sum_exp += __shfl_down_sync(0xffffffff, sum_exp, offset);
    }

    __shared__ float shared_sum_exp;
    if (idx == 0) shared_sum_exp = sum_exp;
    __syncthreads();

    // Compute softmax
    if (idx < output_size) {
        softmax_out[idx] = expf(output[idx] - shared_max) / shared_sum_exp;
    }
}

// Function to initialize data arrays with specific formulas
void initializeData(float* input, float* W1, float* b1, float* W2, float* b2, int input_size, int hidden_size, int output_size) {
    for (int i = 0; i < input_size; ++i) {
        input[i] = i * 5e-10f;
    }
    for (int i = 0; i < hidden_size; ++i) {
        b1[i] = 0.002;
        for (int j = 0; j < input_size; ++j) {
            W1[i * input_size + j] = i * j * 7e-9f;
        }
    }
    for (int i = 0; i < output_size; ++i) {
        b2[i] = 0.002;
        for (int j = 0; j < hidden_size; ++j) {
            W2[i * hidden_size + j] = i * j * 9e-9f;
        }
    }
}

int main() {
    const int input_size = 10000;
    const int hidden_size = 20000;
    const int output_size = 100;

    // Allocate host memory
    float *h_input = new float[input_size];
    float *h_hidden = new float[hidden_size];
    float *h_output = new float[output_size];
    float *h_linear1_weights = new float[input_size * hidden_size];
    float *h_linear1_bias = new float[hidden_size];
    float *h_linear2_weights = new float[hidden_size * output_size];
    float *h_linear2_bias = new float[output_size];

    initializeData(h_input, h_linear1_weights, h_linear1_bias, h_linear2_weights, h_linear2_bias, 
                  input_size, hidden_size, output_size);

    // Allocate device memory
    float *d_input, *d_hidden, *d_output, *d_softmax_out;
    float *d_linear1_weights, *d_linear1_bias;
    float *d_linear2_weights, *d_linear2_bias;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_hidden, hidden_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));
    hipMalloc(&d_linear1_weights, input_size * hidden_size * sizeof(float));
    hipMalloc(&d_linear1_bias, hidden_size * sizeof(float));
    hipMalloc(&d_linear2_weights, hidden_size * output_size * sizeof(float));
    hipMalloc(&d_linear2_bias, output_size * sizeof(float));
    hipMalloc(&d_softmax_out, output_size * sizeof(float));

    // Copy data to device
    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_weights, h_linear1_weights, input_size * hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_bias, h_linear1_bias, hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_weights, h_linear2_weights, hidden_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_bias, h_linear2_bias, output_size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernels with specified configurations
    dim3 mm_gridDim(5000, 1, 1);
    dim3 mm_blockDim(32, 4, 1);
    mm_cuda<<<mm_gridDim, mm_blockDim, input_size * sizeof(float)>>>(d_linear1_weights, d_input, d_hidden, hidden_size, input_size);

    dim3 relu_gridDim(79, 1, 1);
    dim3 relu_blockDim(128, 1, 1);
    ReLU_cuda<<<relu_gridDim, relu_blockDim>>>(d_hidden, d_linear1_bias, hidden_size);

    dim3 addmm_gridDim(25, 1, 4);
    dim3 addmm_blockDim(34, 4, 1);
    addmm_cuda<<<addmm_gridDim, addmm_blockDim, (input_size + hidden_size) * sizeof(float)>>>(d_linear2_weights, d_hidden, d_linear2_bias, d_output, output_size, hidden_size);

    dim3 softmax_gridDim(1, 1, 1);
    dim3 softmax_blockDim(64, 1, 1);
    softmax_1_cuda<<<softmax_gridDim, softmax_blockDim>>>(d_output, d_softmax_out, output_size);

    // Copy results back to host
    hipMemcpy(h_output, d_softmax_out, output_size * sizeof(float), hipMemcpyDeviceToHost);

    // Print results
    std::cout << "\nOutput: " << std::endl;
    for (int i = 0; i < output_size; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    // Cleanup
    delete[] h_input;
    delete[] h_hidden;
    delete[] h_output;
    delete[] h_linear1_weights;
    delete[] h_linear1_bias;
    delete[] h_linear2_weights;
    delete[] h_linear2_bias;

    hipFree(d_input);
    hipFree(d_hidden);
    hipFree(d_output);
    hipFree(d_linear1_weights);
    hipFree(d_linear1_bias);
    hipFree(d_linear2_weights);
    hipFree(d_linear2_bias);
    hipFree(d_softmax_out);

    return 0;
}
