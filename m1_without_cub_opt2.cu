#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cfloat>
#include <sys/time.h>

#define BLOCK_SIZE 256

// CUDA Kernel for fused matrix-vector multiplication with bias and Tanh activation
__global__ void fused_tanh_0_cuda(const float* mat, const float* vec, const float* bias, float* output, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        float mul = bias[row];
        for (int col = 0; col < cols; ++col) {
            mul += mat[row * cols + col] * vec[col];
        }
        output[row] = tanhf(mul); // Apply Tanh activation
    }
}

// CUDA Kernel for matrix-vector multiplication with bias
__global__ void addmm_1_cuda(const float* mat, const float* vec, const float* bias, float* output, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        float sum = bias[row];
        for (int col = 0; col < cols; ++col) {
            sum += mat[row * cols + col] * vec[col];
        }
        output[row] = sum; // Store the result
    }
}

// CUDA Kernel for Softmax computation using parallel reduction
__global__ void softmax_2_cuda(float* output, float* softmax_out, int output_size) {
    extern __shared__ float shared_data[];
    
    int tid = threadIdx.x;
    float data = output[tid];
    
    // Load data into shared memory
    shared_data[tid] = data;
    __syncthreads();
    
    // Parallel reduction to find the maximum
    for (int stride = output_size / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] = fmaxf(shared_data[tid], shared_data[tid + stride]);
        }
        __syncthreads();
    }
    
    float max_val = shared_data[0];
    
    // Compute exponentials and their sum
    float exp_data = expf(data - max_val);
    shared_data[tid] = exp_data;
    __syncthreads();
    
    // Parallel reduction to compute the sum
    for (int stride = output_size / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }
    
    float sum_exp = shared_data[0];
    
    // Compute softmax
    softmax_out[tid] = exp_data / sum_exp;
}

void initializeData(float* input, float* W1, float* b1, float* W2, float* b2, int input_size, int hidden_size, int output_size) {
    for (int i = 0; i < input_size; ++i) {
        input[i] = 0.0001 * i + 0.001;
    }
    for (int i = 0; i < hidden_size; ++i) {
        b1[i] = 0.0005;
        for (int j = 0; j < input_size; ++j) {
            W1[i * input_size + j] = 0.0001 * i * j;
        }
    }
    for (int i = 0; i < output_size; ++i) {
        b2[i] = 0.0005;
        for (int j = 0; j < hidden_size; ++j) {
            W2[i * hidden_size + j] = 0.0001 * i * j;
        }
    }
}

int main() {
    const int input_size = 100;
    const int hidden_size = 200;
    const int output_size = 10;
    float* d_softmax_out;

    // Allocate host memory
    float* h_input = new float[input_size];
    float* h_hidden = new float[hidden_size];
    float* h_output = new float[output_size];

    float* h_linear1_weights = new float[input_size * hidden_size];
    float* h_linear1_bias = new float[hidden_size];
    float* h_linear2_weights = new float[hidden_size * output_size];
    float* h_linear2_bias = new float[output_size];

    // Initialize data
    initializeData(h_input, h_linear1_weights, h_linear1_bias, h_linear2_weights, h_linear2_bias, input_size, hidden_size, output_size);

    // Allocate device memory
    float *d_input, *d_hidden, *d_output;
    float *d_linear1_weights, *d_linear1_bias;
    float *d_linear2_weights, *d_linear2_bias;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_hidden, hidden_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));
    hipMalloc(&d_linear1_weights, input_size * hidden_size * sizeof(float));
    hipMalloc(&d_linear1_bias, hidden_size * sizeof(float));
    hipMalloc(&d_linear2_weights, hidden_size * output_size * sizeof(float));
    hipMalloc(&d_linear2_bias, output_size * sizeof(float));
    hipMalloc((void**)&d_softmax_out, output_size * sizeof(float));

    // Copy data to device
    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_weights, h_linear1_weights, input_size * hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_bias, h_linear1_bias, hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_weights, h_linear2_weights, hidden_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_bias, h_linear2_bias, output_size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernels
    dim3 fused_tanh_gridDim(7, 1, 1);
    dim3 fused_tanh_blockDim(64, 1, 1);
    fused_tanh_0_cuda<<<fused_tanh_gridDim, fused_tanh_blockDim>>>(d_linear1_weights, d_input, d_linear1_bias, d_hidden, hidden_size, input_size);

    dim3 addmm_gridDim(1, 1, 1);
    dim3 addmm_blockDim(32, 1, 1);
    addmm_1_cuda<<<addmm_gridDim, addmm_blockDim>>>(d_linear2_weights, d_hidden, d_linear2_bias, d_output, output_size, hidden_size);

    dim3 softmax_gridDim(1, 1, 1);
    dim3 softmax_blockDim(64, 1, 1);
    softmax_2_cuda<<<softmax_gridDim, softmax_blockDim, output_size * sizeof(float)>>>(d_output, d_softmax_out, output_size);

    // Copy results back to host
    hipMemcpy(h_output, d_softmax_out, output_size * sizeof(float), hipMemcpyDeviceToHost);

    // Output the results
    std::cout << "\nOutput: " << std::endl;
    for (int i = 0; i < output_size; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    // Clean up
    delete[] h_input;
    delete[] h_hidden;
    delete[] h_output;
    delete[] h_linear1_weights;
    delete[] h_linear1_bias;
    delete[] h_linear2_weights;
    delete[] h_linear2_bias;

    hipFree(d_input);
    hipFree(d_hidden);
    hipFree(d_output);
    hipFree(d_linear1_weights);
    hipFree(d_linear1_bias);
    hipFree(d_linear2_weights);
    hipFree(d_linear2_bias);

    return 0;
}