#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cfloat>

#define BLOCK_SIZE 256 // Optimized block size for most CUDA architectures

// CUDA Kernel for matrix-vector multiplication with bias
__global__ void mm_cuda(const float* mat, const float* vec, float* output, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        float sum = 0.0f;
        for (int col = 0; col < cols; ++col) {
            sum += mat[row * cols + col] * vec[col];
        }
        output[row] = sum;
    }
}

// CUDA Kernel for Tanh activation
__global__ void Tanh_cuda(float* data, const float* bias, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = tanhf(data[idx] + bias[idx]);
    }
}

// CUDA Kernel for matrix-vector addition with bias
__global__ void addmm_cuda(const float* mat, const float* vec, const float* bias, float* output, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        float sum = bias[row];
        for (int col = 0; col < cols; ++col) {
            sum += mat[row * cols + col] * vec[col];
        }
        output[row] = sum;
    }
}

// CUDA Kernel for Softmax computation
__global__ void softmax_1_cuda(float* output, float* softmax_out, int output_size) {
    __shared__ float max_val;
    __shared__ float sum_exp;

    if (threadIdx.x == 0) {
        max_val = -FLT_MAX;
        sum_exp = 0.0f;
        for (int i = 0; i < output_size; ++i) {
            max_val = max(max_val, output[i]);
        }
    }
    __syncthreads();

    float exp_val = expf(output[threadIdx.x] - max_val);
    atomicAdd(&sum_exp, exp_val);
    __syncthreads();

    if (threadIdx.x < output_size) {
        softmax_out[threadIdx.x] = expf(output[threadIdx.x] - max_val) / sum_exp;
    }
}

// Function to initialize data arrays
void initializeData(float* input, float* W1, float* b1, float* W2, float* b2, int input_size, int hidden_size, int output_size) {
    for (int i = 0; i < input_size; ++i) {
        input[i] = 0.0001f * i + 0.001f;
    }

    for (int i = 0; i < hidden_size; ++i) {
        b1[i] = 0.0005f;
        for (int j = 0; j < input_size; ++j) {
            W1[i * input_size + j] = 0.0001f * i * j;
        }
    }

    for (int i = 0; i < output_size; ++i) {
        b2[i] = 0.0005f;
        for (int j = 0; j < hidden_size; ++j) {
            W2[i * hidden_size + j] = 0.0001f * i * j;
        }
    }
}

int main() {
    const int input_size = 100;
    const int hidden_size = 200;
    const int output_size = 10;

    float *h_input = new float[input_size];
    float *h_hidden = new float[hidden_size];
    float *h_output = new float[output_size];
    float *h_linear1_weights = new float[input_size * hidden_size];
    float *h_linear1_bias = new float[hidden_size];
    float *h_linear2_weights = new float[hidden_size * output_size];
    float *h_linear2_bias = new float[output_size];

    initializeData(h_input, h_linear1_weights, h_linear1_bias, h_linear2_weights, h_linear2_bias, input_size, hidden_size, output_size);

    float *d_input, *d_hidden, *d_output;
    float *d_linear1_weights, *d_linear1_bias;
    float *d_linear2_weights, *d_linear2_bias, *d_softmax_out;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_hidden, hidden_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMalloc(&d_linear1_weights, input_size * hidden_size * sizeof(float));
    hipMalloc(&d_linear1_bias, hidden_size * sizeof(float));
    hipMalloc(&d_linear2_weights, hidden_size * output_size * sizeof(float));
    hipMalloc(&d_linear2_bias, output_size * sizeof(float));
    hipMalloc(&d_softmax_out, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_weights, h_linear1_weights, input_size * hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_bias, h_linear1_bias, hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_weights, h_linear2_weights, hidden_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_bias, h_linear2_bias, output_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 gridDim_hidden((hidden_size + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 blockDim_hidden(BLOCK_SIZE);
    mm_cuda<<<gridDim_hidden, blockDim_hidden>>>(d_linear1_weights, d_input, d_hidden, hidden_size, input_size);
    Tanh_cuda<<<gridDim_hidden, blockDim_hidden>>>(d_hidden, d_linear1_bias, hidden_size);

    dim3 gridDim_output((output_size + BLOCK_SIZE - 1) / BLOCK_SIZE);
    addmm_cuda<<<gridDim_output, blockDim_hidden>>>(d_linear2_weights, d_hidden, d_linear2_bias, d_output, output_size, hidden_size);
    softmax_1_cuda<<<1, output_size>>>(d_output, d_softmax_out, output_size);

    hipMemcpy(h_output, d_softmax_out, output_size * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Output: \n";
    for (int i = 0; i < output_size; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    delete[] h_input;
    delete[] h_hidden;
    delete[] h_output;
    delete[] h_linear1_weights;
    delete[] h_linear1_bias;
    delete[] h_linear2_weights;
    delete[] h_linear2_bias;

    hipFree(d_input);
    hipFree(d_hidden);
    hipFree(d_output);
    hipFree(d_linear1_weights);
    hipFree(d_linear1_bias);
    hipFree(d_linear2_weights);
    hipFree(d_linear2_bias);
    hipFree(d_softmax_out);

    return 0;
}
