#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cfloat>
#include <sys/time.h>

// CUDA Kernel for fused matrix-vector multiplication with bias and ReLU activation
__global__ void fused_relu_0_cuda(const float* mat, const float* vec, const float* bias, float* output, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        float mul = bias[row];
        for (int col = 0; col < cols; ++col) {
            mul += mat[row * cols + col] * vec[col];
        }
        output[row] = mul > 0.0f ? mul : 0.0f; // Apply ReLU activation
    }
}

// CUDA Kernel for matrix-vector multiplication with bias
__global__ void addmm_1_cuda(const float* mat, const float* vec, const float* bias, float* output, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        float sum = bias[row];
        for (int col = 0; col < cols; ++col) {
            sum += mat[row * cols + col] * vec[col];
        }
        output[row] = sum; // Store the result
    }
}

// CUDA Kernel for Softmax computation
__global__ void softmax_2_cuda(float* output, float* softmax_out, int output_size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < output_size) {
        float max_val = output[0];
        for (int i = 1; i < output_size; ++i) {
            max_val = max(max_val, output[i]); // Find the maximum value
        }
        float sum_exp = 0.0f;
        for (int i = 0; i < output_size; ++i) {
            sum_exp += expf(output[i] - max_val); // Compute the exponential sum
        }
        softmax_out[idx] = expf(output[idx] - max_val) / sum_exp; // Normalize
    }
}

void initializeData(float* input, float* W1, float* b1, float* W2, float* b2, int input_size, int hidden_size, int output_size) {
    for (int i = 0; i < input_size; ++i) {
        input[i] = 0.0001 * i + 0.001;
    }
    for (int i = 0; i < hidden_size; ++i) {
        b1[i] = 0.0005;
        for (int j = 0; j < input_size; ++j) {
            W1[i * input_size + j] = 0.0001 * i * j;
        }
    }
    for (int i = 0; i < output_size; ++i) {
        b2[i] = 0.0005;
        for (int j = 0; j < hidden_size; ++j) {
            W2[i * hidden_size + j] = 0.0001 * i * j;
        }
    }
}

int main() {
    const int input_size = 10000;
    const int hidden_size = 20000;
    const int output_size = 100;
    float* d_softmax_out;

    // Allocate host memory
    float* h_input = new float[input_size];
    float* h_hidden = new float[hidden_size];
    float* h_output = new float[output_size];

    float* h_linear1_weights = new float[input_size * hidden_size];
    float* h_linear1_bias = new float[hidden_size];
    float* h_linear2_weights = new float[hidden_size * output_size];
    float* h_linear2_bias = new float[output_size];

    // Initialize data
    initializeData(h_input, h_linear1_weights, h_linear1_bias, h_linear2_weights, h_linear2_bias, input_size, hidden_size, output_size);

    // Allocate device memory
    float *d_input, *d_hidden, *d_output;
    float *d_linear1_weights, *d_linear1_bias;
    float *d_linear2_weights, *d_linear2_bias;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_hidden, hidden_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));
    hipMalloc(&d_linear1_weights, input_size * hidden_size * sizeof(float));
    hipMalloc(&d_linear1_bias, hidden_size * sizeof(float));
    hipMalloc(&d_linear2_weights, hidden_size * output_size * sizeof(float));
    hipMalloc(&d_linear2_bias, output_size * sizeof(float));
    hipMalloc((void**)&d_softmax_out, output_size * sizeof(float));

    // Copy data to device
    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_weights, h_linear1_weights, input_size * hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_bias, h_linear1_bias, hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_weights, h_linear2_weights, hidden_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_bias, h_linear2_bias, output_size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernels
    dim3 fused_relu_gridDim((hidden_size + 255) / 256);
    dim3 fused_relu_blockDim(256);
    fused_relu_0_cuda<<<fused_relu_gridDim, fused_relu_blockDim>>>(d_linear1_weights, d_input, d_linear1_bias, d_hidden, hidden_size, input_size);

    dim3 addmm_gridDim((output_size + 255) / 256);
    dim3 addmm_blockDim(256);
    addmm_1_cuda<<<addmm_gridDim, addmm_blockDim>>>(d_linear2_weights, d_hidden, d_linear2_bias, d_output, output_size, hidden_size);

    dim3 softmax_gridDim((output_size + 255) / 256);
    dim3 softmax_blockDim(256);
    softmax_2_cuda<<<softmax_gridDim, softmax_blockDim>>>(d_output, d_softmax_out, output_size);

    // Copy results back to host
    hipMemcpy(h_output, d_softmax_out, output_size * sizeof(float), hipMemcpyDeviceToHost);

    // Output the results
    std::cout << "\nOutput: " << std::endl;
    for (int i = 0; i < output_size; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    // Clean up
    delete[] h_input;
    delete[] h_hidden;
    delete[] h_output;
    delete[] h_linear1_weights;
    delete[] h_linear1_bias;
    delete[] h_linear2_weights;
    delete[] h_linear2_bias;

    hipFree(d_input);
    hipFree(d_hidden);
    hipFree(d_output);
    hipFree(d_linear1_weights);
    hipFree(d_linear1_bias);
    hipFree(d_linear2_weights);
    hipFree(d_linear2_bias);

    return 0;
}
