#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cfloat>
#include <sys/time.h>

#define BLOCK_SIZE 256

// Optimized CUDA Kernel for matrix-vector multiplication with bias
__global__ void mm_cuda(const float* mat, const float* vec, float* output, int rows, int cols) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= rows) return;

    float sum = 0.0f;
    for (int col = 0; col < cols; ++col) {
        sum += mat[tid * cols + col] * vec[col];
    }
    output[tid] = sum;
}

// Optimized CUDA Kernel for Tanh activation
__global__ void Tanh_cuda(float* data, const float* bias, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= size) return;
    data[tid] = tanhf(data[tid] + bias[tid]);
}

// Optimized CUDA Kernel for matrix-vector addition with bias
__global__ void addmm_cuda(const float* mat, const float* vec, const float* bias, float* output, int rows, int cols) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= rows) return;

    float sum = bias[tid];
    for (int col = 0; col < cols; ++col) {
        sum += mat[tid * cols + col] * vec[col];
    }
    output[tid] = sum;
}

// Optimized CUDA Kernel for Softmax computation
__global__ void softmax_1_cuda(float* output, float* softmax_out, int output_size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= output_size) return;

    float max_val = output[0];
    for (int i = 1; i < output_size; ++i) {
        if (output[i] > max_val) {
            max_val = output[i];
        }
    }

    float sum_exp = 0.0f;
    for (int i = 0; i < output_size; ++i) {
        sum_exp += expf(output[i] - max_val);
    }

    softmax_out[tid] = expf(output[tid] - max_val) / sum_exp;
}

void initializeData(float* input, float* W1, float* b1, float* W2, float* b2, int input_size, int hidden_size, int output_size) {
    for (int i = 0; i < input_size; ++i) {
        input[i] = 0.0001 * i + 0.001;
    }

    for (int i = 0; i < hidden_size; ++i) {
        b1[i] = 0.0005;
        for (int j = 0; j < input_size; ++j) {
            W1[i * input_size + j] = 0.0001 * i * j;
        }
    }

    for (int i = 0; i < output_size; ++i) {
        b2[i] = 0.0005;
        for (int j = 0; j < hidden_size; ++j) {
            W2[i * hidden_size + j] = 0.0001 * i * j;
        }
    }
}

int main() {
    const int input_size = 100;
    const int hidden_size = 200;
    const int output_size = 10;

    float *h_input = new float[input_size];
    float *h_hidden = new float[hidden_size];
    float *h_output = new float[output_size];
    float *h_linear1_weights = new float[input_size * hidden_size];
    float *h_linear1_bias = new float[hidden_size];
    float *h_linear2_weights = new float[hidden_size * output_size];
    float *h_linear2_bias = new float[output_size];

    initializeData(h_input, h_linear1_weights, h_linear1_bias, h_linear2_weights, h_linear2_bias, input_size, hidden_size, output_size);

    float *d_input, *d_hidden, *d_output;
    float *d_linear1_weights, *d_linear1_bias;
    float *d_linear2_weights, *d_linear2_bias;
    float *d_softmax_out;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_hidden, hidden_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));
    hipMalloc(&d_linear1_weights, input_size * hidden_size * sizeof(float));
    hipMalloc(&d_linear1_bias, hidden_size * sizeof(float));
    hipMalloc(&d_linear2_weights, hidden_size * output_size * sizeof(float));
    hipMalloc(&d_linear2_bias, output_size * sizeof(float));
    hipMalloc(&d_softmax_out, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_weights, h_linear1_weights, input_size * hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_bias, h_linear1_bias, hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_weights, h_linear2_weights, hidden_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_bias, h_linear2_bias, output_size * sizeof(float), hipMemcpyHostToDevice);

    // Launch mm_cuda kernel
    int threadsPerBlock = 256;
    int blocks = (hidden_size + threadsPerBlock - 1) / threadsPerBlock;
    mm_cuda<<<blocks, threadsPerBlock>>>(d_linear1_weights, d_input, d_hidden, hidden_size, input_size);

    // Launch Tanh_cuda kernel
    blocks = (hidden_size + threadsPerBlock - 1) / threadsPerBlock;
    Tanh_cuda<<<blocks, threadsPerBlock>>>(d_hidden, d_linear1_bias, hidden_size);

    // Launch addmm_cuda kernel
    blocks = (output_size + threadsPerBlock - 1) / threadsPerBlock;
    addmm_cuda<<<blocks, threadsPerBlock>>>(d_linear2_weights, d_hidden, d_linear2_bias, d_output, output_size, hidden_size);

    // Launch softmax_1_cuda kernel
    blocks = (output_size + threadsPerBlock - 1) / threadsPerBlock;
    softmax_1_cuda<<<blocks, threadsPerBlock>>>(d_output, d_softmax_out, output_size);

    hipMemcpy(h_output, d_softmax_out, output_size * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "\nOutput: " << std::endl;
    for (int i = 0; i < output_size; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    delete[] h_input;
    delete[] h_hidden;
    delete[] h_output;
    delete[] h_linear1_weights;
    delete[] h_linear1_bias;
    delete[] h_linear2_weights;
    delete[] h_linear2_bias;

    hipFree(d_input);
    hipFree(d_hidden);
    hipFree(d_output);
    hipFree(d_linear1_weights);
    hipFree(d_linear1_bias);
    hipFree(d_linear2_weights);
    hipFree(d_linear2_bias);
    hipFree(d_softmax_out);

    return 0;
}
