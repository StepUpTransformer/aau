#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cfloat>
#include <sys/time.h>

#define BLOCK_SIZE 256

__global__ void mm_cuda(const float* mat, const float* vec, float* output, int rows, int cols) { 
    int idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; 

    if (idx < rows) {
        float mul = 0.0f;
        for (int col = 0; col < cols; ++col) {
            mul += mat[idx * cols + col] * vec[col];
        }
        output[idx] = mul;
    }
}

__global__ void Tanh_cuda(float* data, const float* bias, int size) {
    int idx = threadIdx.x;
    int stride = blockDim.x;

    for (int i = idx; i < size; i += stride) {
        data[i] = tanhf(data[i] + bias[i]);
    }
}

__global__ void addmm_cuda(const float* mat, const float* vec, const float* bias, float* output, int rows, int cols) { 
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows) {
        float sum = bias[row];
        for (int col = 0; col < cols; ++col) {
            sum += mat[row * cols + col] * vec[col];
        }
        output[row] = sum;
    }
}

__global__ void softmax_1_cuda(float* output, float* softmax_out, int output_size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < output_size) {
        extern __shared__ float shared_data[];
        shared_data[threadIdx.x] = output[idx];
        __syncthreads();

        float max_val = shared_data[0];
        for (int i = 1; i < output_size; ++i) {
            max_val = fmaxf(max_val, shared_data[i]);
        }
        __syncthreads();

        float sum_exp = 0.0f;
        for (int i = 0; i < output_size; ++i) { 
            sum_exp += expf(shared_data[i] - max_val);
        }
        __syncthreads();

        softmax_out[idx] = expf(shared_data[idx] - max_val) / sum_exp;
    }
}

void initializeData(float* input, float* W1, float* b1, float* W2, float* b2, int input_size, int hidden_size, int output_size) { 
    for (int i = 0; i < input_size; ++i) {
        input[i] = 0.0001 * i + 0.001;
    }

    for (int i = 0; i < hidden_size; ++i) {
        b1[i] = 0.0005;
        for (int j = 0; j < input_size; ++j) {
            W1[i * input_size + j] = 0.0001 * i * j;
        }
    }

    for (int i = 0; i < output_size; ++i) {
        b2[i] = 0.0005;
        for (int j = 0; j < hidden_size; ++j) {
            W2[i * hidden_size + j] = 0.0001 * i * j;
        }
    }
}

int main() {
    const int input_size = 100; 
    const int hidden_size = 200; 
    const int output_size = 10; 
    float *d_softmax_out;

    float *h_input = new float[input_size]; 
    float *h_hidden = new float[hidden_size];
    float *h_output = new float[output_size];
    float *h_linear1_weights = new float[input_size * hidden_size];
    float *h_linear1_bias = new float[hidden_size];
    float *h_linear2_weights = new float[hidden_size * output_size];
    float *h_linear2_bias = new float[output_size];

    initializeData(h_input, h_linear1_weights, h_linear1_bias, h_linear2_weights, h_linear2_bias, input_size, hidden_size, output_size);

    hipMalloc(&d_input, input_size * sizeof(float)); 
    hipMalloc(&d_hidden, hidden_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));
    hipMalloc(&d_linear1_weights, input_size * hidden_size * sizeof(float));
    hipMalloc(&d_linear1_bias, hidden_size * sizeof(float));
    hipMalloc(&d_linear2_weights, hidden_size * output_size * sizeof(float));
    hipMalloc(&d_linear2_bias, output_size * sizeof(float));
    hipMalloc(&d_softmax_out, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_weights, h_linear1_weights, input_size * hidden_size * sizeof(float), hipMemcpyHostToDevice); 
    hipMemcpy(d_linear1_bias, h_linear1_bias, hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_weights, h_linear2_weights, hidden_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_bias, h_linear2_bias, output_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 mm_gridDim(25, 1, 1); 
    dim3 mm_blockDim(8, 8, 1); 
    mm_cuda<<<mm_gridDim, mm_blockDim>>>(d_linear1_weights, d_input, d_hidden, hidden_size, input_size);

    dim3 Tanh_gridDim(1, 1, 1); 
    dim3 Tanh_blockDim(128, 1, 1); 
    Tanh_cuda<<<Tanh_gridDim, Tanh_blockDim>>>(d_hidden, d_linear1_bias, hidden_size);

    dim3 addmm_gridDim(2, 1, 1); 
    dim3 addmm_blockDim(16, 8, 1); 
    addmm_cuda<<<addmm_gridDim, addmm_blockDim>>>(d_linear2_weights, d_hidden, d_linear2_bias, d_output, output_size, hidden_size);

    dim3 softmax_gridDim(1, 1, 1); 
    dim3 softmax_blockDim(64, 1, 1); 
    float* shared_mem = NULL;
    softmax_1_cuda<<<softmax_gridDim, softmax_blockDim, output_size * sizeof(float)>>>(d_output, d_softmax_out, output_size);

    hipMemcpy(h_output, d_softmax_out, output_size * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "\nOutput: " << std::endl;
    for (int i = 0; i < output_size; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    delete[] h_input;
    delete[] h_hidden;
    delete[] h_output;
    delete[] h_linear1_weights;
    delete[] h_linear1_bias;
    delete[] h_linear2_weights;
    delete[] h_linear2_bias;

    hipFree(d_input);
    hipFree(d_hidden);
    hipFree(d_output);
    hipFree(d_linear1_weights);
    hipFree(d_linear1_bias);
    hipFree(d_linear2_weights);
    hipFree(d_linear2_bias);

    return 0;
}
