#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cfloat>
#include <sys/time.h>

#define BLOCK_SIZE 256

// CUDA Kernel for matrix-vector multiplication with bias
__global__ void mm_cuda(const float* mat, const float* vec, float* output, int rows, int cols) { 
    int idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; 
    if (idx < rows) {
        float mul = 0.0f;
        for (int col = 0; col < cols; ++col) {
           mul += mat[idx * cols + col] * vec[col];
        }
        output[idx] = mul;
    }
}

// CUDA Kernel for Tanh activation
__global__ void Tanh_cuda(float* data, const float* bias, int size) {
    int idx = threadIdx.x;
    int stride = blockDim.x;
    for (int i = idx; i < size; i += stride) {
        data[i] = tanhf(data[i] + bias[i]);
    }
}

// CUDA Kernel for matrix-vector multiplication with bias
__global__ void addmm_cuda(const float* mat, const float* vec, const float* bias, float* output, int rows, int cols) { 
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        float sum = bias[row];
        for (int col = 0; col < cols; ++col) {
            sum += mat[row * cols + col] * vec[col];
        }
        output[row] = sum;
    }
}

// CUDA Kernel for Softmax
__global__ void softmax_1_cuda(float* output, float* softmax_out, int output_size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < output_size) {
        float max_val = output[0];
        for (int i = 1; i < output_size; ++i) {
            max_val = max(max_val, output[i]);
        }
        float sum_exp = 0.0f;
        for (int i = 0; i < output_size; ++i) { 
            sum_exp += expf(output[i] - max_val);
        }
        softmax_out[idx] = expf(output[idx] - max_val) / sum_exp;
    }
}

// Other parts of the code remain unchanged...

int main() {
    const int input_size = 100; 
    const int hidden_size = 200; 
    const int output_size = 10; 
    float *d_softmax_out;

    // Allocate host memory
    float *h_input = new float[input_size]; 
    float *h_hidden = new float[hidden_size];
    float *h_output = new float[output_size];
    float *h_linear1_weights = new float[input_size * hidden_size];
    float *h_linear1_bias = new float[hidden_size];
    float *h_linear2_weights = new float[hidden_size * output_size];
    float *h_linear2_bias = new float[output_size];

    // Initialize data
    initializeData(h_input, h_linear1_weights, h_linear1_bias, h_linear2_weights, h_linear2_bias, input_size, hidden_size, output_size);

    // Allocate device memory
    float *d_input, *d_hidden, *d_output;
    float *d_linear1_weights, *d_linear1_bias;
    float *d_linear2_weights, *d_linear2_bias;

    hipMalloc(&d_input, input_size * sizeof(float)); 
    hipMalloc(&d_hidden, hidden_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMalloc(&d_linear1_weights, input_size * hidden_size * sizeof(float));
    hipMalloc(&d_linear1_bias, hidden_size * sizeof(float));
    hipMalloc(&d_linear2_weights, hidden_size * output_size * sizeof(float));
    hipMalloc(&d_linear2_bias, output_size * sizeof(float));
    hipMalloc((void**)&d_softmax_out, output_size * sizeof(float));

    // Copy data to device
    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_weights, h_linear1_weights, input_size * hidden_size * sizeof(float), hipMemcpyHostToDevice); 
    hipMemcpy(d_linear1_bias, h_linear1_bias, hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_weights, h_linear2_weights, hidden_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_bias, h_linear2_bias, output_size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernels
    dim3 mm_gridDim(25, 1, 1);
    dim3 mm_blockDim(8, 8, 1);
    mm_cuda<<<mm_gridDim, mm_blockDim>>>(d_linear1_weights, d_input, d_hidden, hidden_size, input_size);

    dim3 Tanh_gridDim(1, 1, 1);
    dim3 Tanh_blockDim(128, 1, 1);
    Tanh_cuda<<<Tanh_gridDim, Tanh_blockDim>>>(d_hidden, d_linear1_bias, hidden_size);

    dim3 addmm_gridDim(2, 1, 1);
    dim3 addmm_blockDim(16, 8, 1);
    addmm_cuda<<<addmm_gridDim, addmm_blockDim>>>(d_linear2_weights, d_hidden, d_linear2_bias, d_output, output_size, hidden_size);

    dim3 softmax_gridDim(1, 1, 1);
    dim3 softmax_blockDim(64, 1, 1);
    softmax_1_cuda<<<softmax_gridDim, softmax_blockDim>>>(d_output, d_softmax_out, output_size);

    // Copy results back to host
    hipMemcpy(h_output, d_softmax_out, output_size * sizeof(float), hipMemcpyDeviceToHost);

    // Output the results
    std::cout << "\nOutput: " << std::endl;
    for (int i = 0; i < output_size; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    // Clean up
    delete[] h_input;
    delete[] h_hidden;
    delete[] h_output;
    delete[] h_linear1_weights;
    delete[] h_linear1_bias;
    delete[] h_linear2_weights;
    delete[] h_linear2_bias;

    hipFree(d_input);
    hipFree(d_hidden);
    hipFree(d_output);
    hipFree(d_linear1_weights);
    hipFree(d_linear1_bias);
    hipFree(d_linear2_weights);
    hipFree(d_linear2_bias);

    return 0;
}
