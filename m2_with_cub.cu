#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cfloat>
#include <sys/time.h>

// CUDA Kernel for matrix-vector multiplication with bias
__global__ void mm_cuda(const float* mat, const float* vec, float* output, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    if (idx < rows) {
        float mul = 0.0f;
        for (int col = 0; col < cols; ++col) {
            mul += mat[idx * cols + col] * vec[col];
        }
        output[idx] = mul;
    }
}

__global__ void Tanh_cuda(float* data, const float* bias, int size) {
    int idx = threadIdx.x;
    int stride = blockDim.x;

    for (int i = idx; i < size; i += stride) {
        data[i] = tanhf(data[i] + bias[i]);
    }
}

__global__ void addmm_cuda(const float* mat, const float* vec, const float* bias, float* output, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows) {
        float sum = bias[row];
        for (int col = 0; col < cols; ++col) {
            sum += mat[row * cols + col] * vec[col];
        }
        output[row] = sum;
    }
}

__global__ void softmax_1_cuda(float* output, float* softmax_out, int output_size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < output_size) {
        float max_val = output[0];
        for (int i = 1; i < output_size; ++i) {
            max_val = max(max_val, output[i]);
        }

        float sum_exp = 0.0f;
        for (int i = 0; i < output_size; ++i) {
            sum_exp += expf(output[i] - max_val);
        }

        softmax_out[idx] = expf(output[idx] - max_val) / sum_exp;
    }
}

// Function to initialize data arrays with specific formulas
void initializeData(float* input, float* W1, float* b1, float* W2, float* b2, int input_size, int hidden_size, int output_size) {
    for (int i = 0; i < input_size; ++i) {
        input[i] = i * 5e-10f;
    }
    for (int i = 0; i < hidden_size; ++i) {
        b1[i] = 0.002;
        for (int j = 0; j < input_size; ++j) {
            W1[i* input_size + j] = i *j* 7e-9f;
        }
    }
    for (int i = 0; i < output_size; ++i) {
        b2[i] = 0.002;
        for (int j = 0; j < hidden_size; ++j) {
            W2[i* hidden_size + j] = i *j* 9e-9f;
        }
    }
}

int main() {
    const int input_size = 10000;
    const int hidden_size = 20000;
    const int output_size = 100;
    float *d_softmax_out;

    float *h_input = new float[input_size];
    float *h_hidden = new float[hidden_size];
    float *h_output = new float[output_size];
    float *h_linear1_weights = new float[input_size * hidden_size];
    float *h_linear1_bias = new float[hidden_size];
    float *h_linear2_weights = new float[hidden_size * output_size];
    float *h_linear2_bias = new float[output_size];

    initializeData(h_input, h_linear1_weights, h_linear1_bias, h_linear2_weights, h_linear2_bias, input_size, hidden_size, output_size);

    float *d_input, *d_hidden, *d_output;
    float *d_linear1_weights, *d_linear1_bias;
    float *d_linear2_weights, *d_linear2_bias;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_hidden, hidden_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMalloc(&d_linear1_weights, input_size * hidden_size * sizeof(float));
    hipMalloc(&d_linear1_bias, hidden_size * sizeof(float));
    hipMalloc(&d_linear2_weights, hidden_size * output_size * sizeof(float));
    hipMalloc(&d_linear2_bias, output_size * sizeof(float));
    hipMalloc((void**)&d_softmax_out, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_weights, h_linear1_weights, input_size * hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_bias, h_linear1_bias, hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_weights, h_linear2_weights, hidden_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_bias, h_linear2_bias, output_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 mm_gridDim(25, 1, 1);
    dim3 mm_blockDim(8, 8, 1);
    mm_cuda<<<mm_gridDim, mm_blockDim>>>(d_linear1_weights, d_input, d_hidden, hidden_size, input_size);

    dim3 Tanh_gridDim(1, 1, 1);
    dim3 Tanh_blockDim(128, 1, 1);
    Tanh_cuda<<<Tanh_gridDim, Tanh_blockDim>>>(d_hidden, d_linear1_bias, hidden_size);

    dim3 addmm_gridDim(2, 1, 1);
    dim3 addmm_blockDim(16, 8, 1);
    addmm_cuda<<<addmm_gridDim, addmm_blockDim>>>(d_linear2_weights, d_hidden, d_linear2_bias, d_output, output_size, hidden_size);

    dim3 softmax_gridDim(1, 1, 1);
    dim3 softmax_blockDim(64, 1, 1);
    softmax_1_cuda<<<softmax_gridDim, softmax_blockDim>>>(d_output, d_softmax_out, output_size);

    hipMemcpy(h_output, d_softmax_out, output_size * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "\nOutput: " << std::endl;
    for (int i = 0; i < output_size; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    delete[] h_input;
    delete[] h_hidden;
    delete[] h_output;
    delete[] h_linear1_weights;
    delete[] h_linear1_bias;
    delete[] h_linear2_weights;
    delete[] h_linear2_bias;

    hipFree(d_input);
    hipFree(d_hidden);
    hipFree(d_output);
    hipFree(d_linear1_weights);
    hipFree(d_linear1_bias);
    hipFree(d_linear2_weights);
    hipFree(d_linear2_bias);

    return 0;
}
