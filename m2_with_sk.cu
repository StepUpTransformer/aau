#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <sys/time.h>

// CUDA Kernel for matrix-vector multiplication with bias
// Grid size: (5000, 1, 1), Block size: (32, 4, 1)
__global__ void mm_cuda(const float* mat, const float* vec, float* output, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col_offset = threadIdx.y;
    if (row < rows) {
        float mul = 0.0f;
        for (int col = col_offset; col < cols; col += blockDim.y) {
            mul += mat[row * cols + col] * vec[col];
        }
        atomicAdd(&output[row], mul);
    }
}

// CUDA Kernel for ReLU activation
// Grid size: (79, 1, 1), Block size: (128, 1, 1)
__global__ void ReLU_cuda(float* data, const float* bias, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float val = data[idx] + bias[idx];
        data[idx] = val > 0.0f ? val : 0.0f;
    }
}

// CUDA Kernel for matrix-vector addition with bias
// Grid size: (25, 1, 4), Block size: (34, 4, 1)
__global__ void addmm_cuda(const float* mat, const float* vec, const float* bias, float* output, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col_offset = blockIdx.z * blockDim.y + threadIdx.y;
    if (row < rows) {
        float sum = (threadIdx.y == 0) ? bias[row] : 0.0f;
        for (int col = col_offset; col < cols; col += blockDim.y * gridDim.z) {
            sum += mat[row * cols + col] * vec[col];
        }
        atomicAdd(&output[row], sum);
    }
}

// CUDA Kernel for Softmax computation
// Grid size: (1, 1, 1), Block size: (64, 1, 1)
__global__ void softmax_1_cuda(float* output, float* softmax_out, int output_size) {
    __shared__ float max_val;
    __shared__ float sum_exp;

    if (threadIdx.x == 0) {
        max_val = output[0];
        for (int i = 1; i < output_size; ++i) {
            max_val = max(max_val, output[i]);
        }
    }
    __syncthreads();

    float local_sum_exp = 0.0f;
    for (int i = threadIdx.x; i < output_size; i += blockDim.x) {
        local_sum_exp += expf(output[i] - max_val);
    }
    atomicAdd(&sum_exp, local_sum_exp);
    __syncthreads();

    if (threadIdx.x < output_size) {
        softmax_out[threadIdx.x] = expf(output[threadIdx.x] - max_val) / sum_exp;
    }
}

// Function to initialize data arrays with specific formulas
void initializeData(float* input, float* W1, float* b1, float* W2, float* b2, int input_size, int hidden_size, int output_size) {
    for (int i = 0; i < input_size; ++i) {
        input[i] = i * 5e-10f;
    }
    for (int i = 0; i < hidden_size; ++i) {
        b1[i] = 0.002;
        for (int j = 0; j < input_size; ++j) {
            W1[i * input_size + j] = i * j * 7e-9f;
        }
    }
    for (int i = 0; i < output_size; ++i) {
        b2[i] = 0.002;
        for (int j = 0; j < hidden_size; ++j) {
            W2[i * hidden_size + j] = i * j * 9e-9f;
        }
    }
}

int main() {
    const int input_size = 10000;
    const int hidden_size = 20000;
    const int output_size = 100;

    float *d_softmax_out;

    float *h_input = new float[input_size];
    float *h_hidden = new float[hidden_size];
    float *h_output = new float[output_size];
    float *h_linear1_weights = new float[input_size * hidden_size];
    float *h_linear1_bias = new float[hidden_size];
    float *h_linear2_weights = new float[hidden_size * output_size];
    float *h_linear2_bias = new float[output_size];

    initializeData(h_input, h_linear1_weights, h_linear1_bias, h_linear2_weights, h_linear2_bias, input_size, hidden_size, output_size);

    float *d_input, *d_hidden, *d_output;
    float *d_linear1_weights, *d_linear1_bias;
    float *d_linear2_weights, *d_linear2_bias;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_hidden, hidden_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMalloc(&d_linear1_weights, input_size * hidden_size * sizeof(float));
    hipMalloc(&d_linear1_bias, hidden_size * sizeof(float));
    hipMalloc(&d_linear2_weights, hidden_size * output_size * sizeof(float));
    hipMalloc(&d_linear2_bias, output_size * sizeof(float));
    hipMalloc((void**)&d_softmax_out, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_weights, h_linear1_weights, input_size * hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_bias, h_linear1_bias, hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_weights, h_linear2_weights, hidden_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_bias, h_linear2_bias, output_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 mm_gridDim(5000, 1, 1);
    dim3 mm_blockDim(32, 4, 1);
    mm_cuda<<<mm_gridDim, mm_blockDim>>>(d_linear1_weights, d_input, d_hidden, hidden_size, input_size);

    dim3 ReLU_gridDim(79, 1, 1);
    dim3 ReLU_blockDim(128, 1, 1);
    ReLU_cuda<<<ReLU_gridDim, ReLU_blockDim>>>(d_hidden, d_linear1_bias, hidden_size);

    dim3 addmm_gridDim(25, 1, 4);
    dim3 addmm_blockDim(34, 4, 1);
    addmm_cuda<<<addmm_gridDim, addmm_blockDim>>>(d_linear2_weights, d_hidden, d_linear2_bias, d_output, output_size, hidden_size);

    dim3 softmax_gridDim(1, 1, 1);
    dim3 softmax_blockDim(64, 1, 1);
    softmax_1_cuda<<<softmax_gridDim, softmax_blockDim>>>(d_output, d_softmax_out, output_size);

    hipMemcpy(h_output, d_softmax_out, output_size * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "\nOutput: " << std::endl;
    for (int i = 0; i < output_size; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    delete[] h_input;
    delete[] h_hidden;
    delete[] h_output;
    delete[] h_linear1_weights;
    delete[] h_linear1_bias;
    delete[] h_linear2_weights;
    delete[] h_linear2_bias;

    hipFree(d_input);
    hipFree(d_hidden);
    hipFree(d_output);
    hipFree(d_linear1_weights);
    hipFree(d_linear1_bias);
    hipFree(d_linear2_weights);
    hipFree(d_linear2_bias);

    return 0;
}
