#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <sys/time.h>

// CUDA Kernel for matrix-vector multiplication with bias
// Grid size: (5000,1,1), Block size: (32,4,1)
__global__ void mm_cuda(const float* mat, const float* vec, float* output, int rows, int cols) {
    // Calculate 2D thread index using block dimensions
    int row = (blockIdx.x * blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    
    if (row < rows) {
        float mul = 0.0f;
        for (int col = 0; col < cols; ++col) {
            mul += mat[row * cols + col] * vec[col];
        }
        output[row] = mul;
    }
}

// CUDA Kernel for ReLU activation
// Grid size: (79,1,1), Block size: (128,1,1)
__global__ void ReLU_cuda(float* data, const float* bias, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float val = data[idx] + bias[idx];
        data[idx] = val > 0.0f ? val : 0.0f;
    }
}

// CUDA Kernel for matrix-vector addition with bias
// Grid size: (25,1,4), Block size: (34,4,1)
__global__ void addmm_cuda(const float* mat, const float* vec, const float* bias, float* output, int rows, int cols) {
    // Calculate 3D thread index
    int row = (blockIdx.x * blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    row += blockIdx.z * (gridDim.x * blockDim.x * blockDim.y);
    
    if (row < rows) {
        float sum = bias[row];
        for (int col = 0; col < cols; ++col) {
            sum += mat[row * cols + col] * vec[col];
        }
        output[row] = sum;
    }
}

// CUDA Kernel for Softmax computation
// Grid size: (1,1,1), Block size: (64,1,1)
__global__ void softmax_1_cuda(float* output, float* softmax_out, int output_size) {
    int idx = threadIdx.x;
    if (idx < output_size) {
        // Find maximum value
        float max_val = output[0];
        for (int i = 1; i < output_size; ++i) {
            max_val = max(max_val, output[i]);
        }
        
        // Calculate sum of exponentials
        float sum_exp = 0.0f;
        for (int i = 0; i < output_size; ++i) {
            sum_exp += expf(output[i] - max_val);
        }
        
        // Calculate softmax
        softmax_out[idx] = expf(output[idx] - max_val) / sum_exp;
    }
}

// Function to initialize data arrays with specific formulas
void initializeData(float* input, float* W1, float* b1, float* W2, float* b2, int input_size, int hidden_size, int output_size) {
    for (int i = 0; i < input_size; ++i) {
        input[i] = i * 5e-10f;
    }
    for (int i = 0; i < hidden_size; ++i) {
        b1[i] = 0.002;
        for (int j = 0; j < input_size; ++j) {
            W1[i * input_size + j] = i * j * 7e-9f;
        }
    }
    for (int i = 0; i < output_size; ++i) {
        b2[i] = 0.002;
        for (int j = 0; j < hidden_size; ++j) {
            W2[i * hidden_size + j] = i * j * 9e-9f;
        }
    }
}

int main() {
    const int input_size = 10000;
    const int hidden_size = 20000;
    const int output_size = 100;

    // Allocate host memory
    float *h_input = new float[input_size];
    float *h_hidden = new float[hidden_size];
    float *h_output = new float[output_size];
    float *h_linear1_weights = new float[input_size * hidden_size];
    float *h_linear1_bias = new float[hidden_size];
    float *h_linear2_weights = new float[hidden_size * output_size];
    float *h_linear2_bias = new float[output_size];

    initializeData(h_input, h_linear1_weights, h_linear1_bias, h_linear2_weights, h_linear2_bias, 
                  input_size, hidden_size, output_size);

    // Allocate device memory
    float *d_input, *d_hidden, *d_output, *d_softmax_out;
    float *d_linear1_weights, *d_linear1_bias;
    float *d_linear2_weights, *d_linear2_bias;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_hidden, hidden_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));
    hipMalloc(&d_linear1_weights, input_size * hidden_size * sizeof(float));
    hipMalloc(&d_linear1_bias, hidden_size * sizeof(float));
    hipMalloc(&d_linear2_weights, hidden_size * output_size * sizeof(float));
    hipMalloc(&d_linear2_bias, output_size * sizeof(float));
    hipMalloc(&d_softmax_out, output_size * sizeof(float));

    // Copy data to device
    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_weights, h_linear1_weights, input_size * hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_bias, h_linear1_bias, hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_weights, h_linear2_weights, hidden_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_bias, h_linear2_bias, output_size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernels with specified configurations
    dim3 mm_gridDim(5000, 1, 1);
    dim3 mm_blockDim(32, 4, 1);
    mm_cuda<<<mm_gridDim, mm_blockDim>>>(d_linear1_weights, d_input, d_hidden, hidden_size, input_size);

    dim3 relu_gridDim(79, 1, 1);
    dim3 relu_blockDim(128, 1, 1);
    ReLU_cuda<<<relu_gridDim, relu_blockDim>>>(d_hidden, d_linear1_bias, hidden_size);

    dim3 addmm_gridDim(25, 1, 4);
    dim3 addmm_blockDim(34, 4, 1);
    addmm_cuda<<<addmm_gridDim, addmm_blockDim>>>(d_linear2_weights, d_hidden, d_linear2_bias, d_output, output_size, hidden_size);

    dim3 softmax_gridDim(1, 1, 1);
    dim3 softmax_blockDim(64, 1, 1);
    softmax_1_cuda<<<softmax_gridDim, softmax_blockDim>>>(d_output, d_softmax_out, output_size);

    // Copy results back to host
    hipMemcpy(h_output, d_softmax_out, output_size * sizeof(float), hipMemcpyDeviceToHost);

    // Print results
    std::cout << "\nOutput: " << std::endl;
    for (int i = 0; i < output_size; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    // Cleanup
    delete[] h_input;
    delete[] h_hidden;
    delete[] h_output;
    delete[] h_linear1_weights;
    delete[] h_linear1_bias;
    delete[] h_linear2_weights;
    delete[] h_linear2_bias;

    hipFree(d_input);
    hipFree(d_hidden);
    hipFree(d_output);
    hipFree(d_linear1_weights);
    hipFree(d_linear1_bias);
    hipFree(d_linear2_weights);
    hipFree(d_linear2_bias);
    hipFree(d_softmax_out);

    return 0;
}
