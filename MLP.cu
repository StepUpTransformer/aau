#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

// Define the ReLU activation function
__global__ void relu(float* x, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        x[idx] = fmaxf(0.0f, x[idx]);
    }
}

// Define the Softmax function
__global__ void softmax(float* x, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float max_val;
    __shared__ float sum_val;

    // Find max value in the array
    if (idx == 0) max_val = -INFINITY;
    __syncthreads();
    atomicMax(&max_val, x[idx]);
    __syncthreads();

    // Calculate the sum of exponentials
    if (idx == 0) sum_val = 0.0f;
    __syncthreads();
    atomicAdd(&sum_val, expf(x[idx] - max_val));
    __syncthreads();

    // Normalize to get softmax
    if (idx < size) {
        x[idx] = expf(x[idx] - max_val) / sum_val;
    }
}

// Define the Linear layer (dense matrix-vector multiplication)
__global__ void linear(const float* input, const float* weights, const float* bias, float* output, int input_dim, int output_dim) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < output_dim) {
        float sum = bias[row];
        for (int col = 0; col < input_dim; ++col) {
            sum += input[col] * weights[row * input_dim + col];
        }
        output[row] = sum;
    }
}

void randomInit(float* data, int size) {
    for (int i = 0; i < size; ++i) {
        data[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

int main() {
    // Model parameters
    const int input_dim = 10000;
    const int hidden_dim = 20000;
    const int output_dim = 10;

    // Allocate memory for the model
    float *linear1_weights, *linear1_bias, *linear2_weights, *linear2_bias;
    hipMallocManaged(&linear1_weights, input_dim * hidden_dim * sizeof(float));
    hipMallocManaged(&linear1_bias, hidden_dim * sizeof(float));
    hipMallocManaged(&linear2_weights, hidden_dim * output_dim * sizeof(float));
    hipMallocManaged(&linear2_bias, output_dim * sizeof(float));

    // Randomly initialize the model parameters
    randomInit(linear1_weights, input_dim * hidden_dim);
    randomInit(linear1_bias, hidden_dim);
    randomInit(linear2_weights, hidden_dim * output_dim);
    randomInit(linear2_bias, output_dim);

    // Allocate memory for input and output
    float *input, *hidden, *output;
    hipMallocManaged(&input, input_dim * sizeof(float));
    hipMallocManaged(&hidden, hidden_dim * sizeof(float));
    hipMallocManaged(&output, output_dim * sizeof(float));

    // Randomly initialize the input
    randomInit(input, input_dim);

    // Launch the Linear layer, ReLU, and Softmax
    int blockSize = 256;
    int gridSize1 = (hidden_dim + blockSize - 1) / blockSize;
    int gridSize2 = (output_dim + blockSize - 1) / blockSize;

    linear<<<gridSize1, blockSize>>>(input, linear1_weights, linear1_bias, hidden, input_dim, hidden_dim);
    hipDeviceSynchronize();

    relu<<<gridSize1, blockSize>>>(hidden, hidden_dim);
    hipDeviceSynchronize();

    linear<<<gridSize2, blockSize>>>(hidden, linear2_weights, linear2_bias, output, hidden_dim, output_dim);
    hipDeviceSynchronize();

    softmax<<<gridSize2, blockSize>>>(output, output_dim);
    hipDeviceSynchronize();

    // Print the output
    std::cout << "Output: ";
    for (int i = 0; i < output_dim; ++i) {
        std::cout << output[i] << " ";
    }
    std::cout << std::endl;

    // Free allocated memory
    hipFree(linear1_weights);
    hipFree(linear1_bias);
    hipFree(linear2_weights);
    hipFree(linear2_bias);
    hipFree(input);
    hipFree(hidden);
    hipFree(output);

    return 0;
}
