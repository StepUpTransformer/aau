#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cfloat>
#include <sys/time.h>

#define BLOCK_SIZE 256

// CUDA Kernel for fused matrix-vector multiplication with bias and Tanh activation
__global__ void fused_tanh_0_cuda(const float* mat, const float* vec, const float* bias, float* output, int rows, int cols) {
    extern __shared__ float shared_vec[];

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_id = threadIdx.x;

    // Load vector into shared memory
    for (int i = thread_id; i < cols; i += blockDim.x) {
        shared_vec[i] = vec[i];
    }
    __syncthreads();

    if (row < rows) {
        float mul = bias[row];
        for (int col = 0; col < cols; ++col) {
            mul += mat[row * cols + col] * shared_vec[col];
        }
        output[row] = tanhf(mul); // Apply Tanh activation
    }
}

// CUDA Kernel for matrix-vector multiplication with bias
__global__ void addmm_1_cuda(const float* mat, const float* vec, const float* bias, float* output, int rows, int cols) {
    extern __shared__ float shared_vec[];

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_id = threadIdx.x;

    // Load vector into shared memory
    for (int i = thread_id; i < cols; i += blockDim.x) {
        shared_vec[i] = vec[i];
    }
    __syncthreads();

    if (row < rows) {
        float sum = bias[row];
        for (int col = 0; col < cols; ++col) {
            sum += mat[row * cols + col] * shared_vec[col];
        }
        output[row] = sum;
    }
}

// CUDA Kernel for Softmax computation
__global__ void softmax_2_cuda(float* output, float* softmax_out, int output_size) {
    extern __shared__ float shared_data[];
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_id = threadIdx.x;

    float max_val = -FLT_MAX;
    if (idx < output_size) {
        max_val = output[idx];
    }

    // Parallel reduction to find max
    shared_data[thread_id] = max_val;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (thread_id < s) {
            shared_data[thread_id] = fmaxf(shared_data[thread_id], shared_data[thread_id + s]);
        }
        __syncthreads();
    }

    max_val = shared_data[0];

    // Compute exponentials and sum
    float sum_exp = 0.0f;
    if (idx < output_size) {
        sum_exp = expf(output[idx] - max_val);
        shared_data[thread_id] = sum_exp;
    } else {
        shared_data[thread_id] = 0.0f;
    }
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (thread_id < s) {
            shared_data[thread_id] += shared_data[thread_id + s];
        }
        __syncthreads();
    }

    float total_sum = shared_data[0];
    if (idx < output_size) {
        softmax_out[idx] = expf(output[idx] - max_val) / total_sum;
    }
}

// Initialize data (unchanged)
void initializeData(float* input, float* W1, float* b1, float* W2, float* b2, int input_size, int hidden_size, int output_size) {
    for (int i = 0; i < input_size; ++i) {
        input[i] = 0.0001 * i + 0.001;
    }
    for (int i = 0; i < hidden_size; ++i) {
        b1[i] = 0.0005;
        for (int j = 0; j < input_size; ++j) {
            W1[i * input_size + j] = 0.0001 * i * j;
        }
    }
    for (int i = 0; i < output_size; ++i) {
        b2[i] = 0.0005;
        for (int j = 0; j < hidden_size; ++j) {
            W2[i * hidden_size + j] = 0.0001 * i * j;
        }
    }
}

int main() {
    const int input_size = 100;
    const int hidden_size = 200;
    const int output_size = 10;
    float* d_softmax_out;

    // Allocate host memory
    float* h_input = new float[input_size];
    float* h_hidden = new float[hidden_size];
    float* h_output = new float[output_size];

    float* h_linear1_weights = new float[input_size * hidden_size];
    float* h_linear1_bias = new float[hidden_size];
    float* h_linear2_weights = new float[hidden_size * output_size];
    float* h_linear2_bias = new float[output_size];

    // Initialize data
    initializeData(h_input, h_linear1_weights, h_linear1_bias, h_linear2_weights, h_linear2_bias, input_size, hidden_size, output_size);

    // Allocate device memory
    float *d_input, *d_hidden, *d_output;
    float *d_linear1_weights, *d_linear1_bias;
    float *d_linear2_weights, *d_linear2_bias;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_hidden, hidden_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));
    hipMalloc(&d_linear1_weights, input_size * hidden_size * sizeof(float));
    hipMalloc(&d_linear1_bias, hidden_size * sizeof(float));
    hipMalloc(&d_linear2_weights, hidden_size * output_size * sizeof(float));
    hipMalloc(&d_linear2_bias, output_size * sizeof(float));
    hipMalloc((void**)&d_softmax_out, output_size * sizeof(float));

    // Copy data to device
    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_weights, h_linear1_weights, input_size * hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_bias, h_linear1_bias, hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_weights, h_linear2_weights, hidden_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_bias, h_linear2_bias, output_size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernels
    dim3 fused_tanh_gridDim(7, 1, 1);
    dim3 fused_tanh_blockDim(64, 1, 1);
    fused_tanh_0_cuda<<<fused_tanh_gridDim, fused_tanh_blockDim, input_size * sizeof(float)>>>(d_linear1_weights, d_input, d_linear1_bias, d_hidden, hidden_size, input_size);

    dim3 addmm_gridDim(1, 1, 1);
    dim3 addmm_blockDim(32, 1, 1);
    addmm_1_cuda<<<addmm_gridDim, addmm_blockDim, hidden_size * sizeof(float)>>>(d_linear2_weights, d_hidden, d_linear2_bias, d_output, output_size, hidden_size);

    dim3 softmax_gridDim(1, 1, 1);
    dim3 softmax_blockDim(64, 1, 1);
    softmax_2_cuda<<<softmax_gridDim, softmax_blockDim, output_size * sizeof(float)>>>(d_output, d_softmax_out, output_size);

    // Copy results back to host
    hipMemcpy(h_output, d_softmax_out, output_size * sizeof(float), hipMemcpyDeviceToHost);

    // Output the results
    std::cout << "\nOutput: " << std::endl;
    for (int i = 0; i < output_size; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    // Clean up
    delete[] h_input;
    delete[] h_hidden;
    delete[] h_output;
    delete[] h_linear1_weights;
    delete[] h_linear1_bias;
    delete[] h_linear2_weights;
    delete[] h_linear2_bias;

    hipFree(d_input);
    hipFree(d_hidden);
    hipFree(d_output);
    hipFree(d_linear1_weights);
    hipFree(d_linear1_bias);
    hipFree(d_linear2_weights);
    hipFree(d_linear2_bias);

    return 0;
}

