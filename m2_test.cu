#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <sys/time.h>

// CUDA Kernel for matrix-vector multiplication with bias
__global__ void mm_cuda(const float* mat, const float* vec, float* output, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows) {
        float mul = 0.0f;
        for (int col = 0; col < cols; ++col) {
            mul += mat[idx * cols + col] * vec[col];
        }
        output[idx] = mul;
    }
}

// CUDA Kernel for ReLU activation
__global__ void ReLU_cuda(float* data, const float* bias, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float val = data[idx] + bias[idx];
        data[idx] = val > 0.0f ? val : 0.0f;
    }
}

// CUDA Kernel for matrix-vector addition with bias
__global__ void addmm_cuda(const float* mat, const float* vec, const float* bias, float* output, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        float sum = bias[row];
        for (int col = 0; col < cols; ++col) {
            sum += mat[row * cols + col] * vec[col];
        }
        output[row] = sum;
    }
}

// CUDA Kernel for Softmax computation
__global__ void softmax_1_cuda(float* output, float* softmax_out, int output_size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < output_size) {
        float max_val = output[0];
        for (int i = 1; i < output_size; ++i) {
            max_val = max(max_val, output[i]);
        }
        float sum_exp = 0.0f;
        for (int i = 0; i < output_size; ++i) {
            sum_exp += expf(output[i] - max_val);
        }
        softmax_out[idx] = expf(output[idx] - max_val) / sum_exp;
    }
}

// Function to initialize data arrays with specific formulas
void initializeData(float* input, float* W1, float* b1, float* W2, float* b2, int input_size, int hidden_size, int output_size) {
    for (int i = 0; i < input_size; ++i) {
        input[i] = 0.0001 * i + 0.001;
    }
    for (int i = 0; i < hidden_size; ++i) {
        b1[i] = 0.0005;
        for (int j = 0; j < input_size; ++j) {
            W1[i * input_size + j] = 0.0001 * i * j;
        }
    }
    for (int i = 0; i < output_size; ++i) {
        b2[i] = 0.0005;
        for (int j = 0; j < hidden_size; ++j) {
            W2[i * hidden_size + j] = 0.0001 * i * j;
        }
    }
}

int main() {
    const int input_size = 10000;
    const int hidden_size = 20000;
    const int output_size = 100;

    float *d_softmax_out;

    float *h_input = new float[input_size];
    float *h_hidden = new float[hidden_size];
    float *h_output = new float[output_size];
    float *h_linear1_weights = new float[input_size * hidden_size];
    float *h_linear1_bias = new float[hidden_size];
    float *h_linear2_weights = new float[hidden_size * output_size];
    float *h_linear2_bias = new float[output_size];

    initializeData(h_input, h_linear1_weights, h_linear1_bias, h_linear2_weights, h_linear2_bias, input_size, hidden_size, output_size);

    float *d_input, *d_hidden, *d_output;
    float *d_linear1_weights, *d_linear1_bias;
    float *d_linear2_weights, *d_linear2_bias;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_hidden, hidden_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMalloc(&d_linear1_weights, input_size * hidden_size * sizeof(float));
    hipMalloc(&d_linear1_bias, hidden_size * sizeof(float));
    hipMalloc(&d_linear2_weights, hidden_size * output_size * sizeof(float));
    hipMalloc(&d_linear2_bias, output_size * sizeof(float));
    hipMalloc((void**)&d_softmax_out, output_size * sizeof(float));

    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_weights, h_linear1_weights, input_size * hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear1_bias, h_linear1_bias, hidden_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_weights, h_linear2_weights, hidden_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_linear2_bias, h_linear2_bias, output_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 mm_gridDim((hidden_size + 255) / 256);
    dim3 mm_blockDim(256);
    mm_cuda<<<mm_gridDim, mm_blockDim>>>(d_linear1_weights, d_input, d_hidden, hidden_size, input_size);

    dim3 ReLU_gridDim((hidden_size + 255) / 256);
    dim3 ReLU_blockDim(256);
    ReLU_cuda<<<ReLU_gridDim, ReLU_blockDim>>>(d_hidden, d_linear1_bias, hidden_size);

    dim3 addmm_gridDim((output_size + 255) / 256);
    dim3 addmm_blockDim(256);
    addmm_cuda<<<addmm_gridDim, addmm_blockDim>>>(d_linear2_weights, d_hidden, d_linear2_bias, d_output, output_size, hidden_size);

    dim3 softmax_gridDim((output_size + 255) / 256);
    dim3 softmax_blockDim(256);
    softmax_1_cuda<<<softmax_gridDim, softmax_blockDim>>>(d_output, d_softmax_out, output_size);

    hipMemcpy(h_output, d_softmax_out, output_size * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "\nOutput: " << std::endl;
    for (int i = 0; i < output_size; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    delete[] h_input;
    delete[] h_hidden;
    delete[] h_output;
    delete[] h_linear1_weights;
    delete[] h_linear1_bias;
    delete[] h_linear2_weights;
    delete[] h_linear2_bias;

    hipFree(d_input);
    hipFree(d_hidden);
    hipFree(d_output);
    hipFree(d_linear1_weights);
    hipFree(d_linear1_bias);
    hipFree(d_linear2_weights);
    hipFree(d_linear2_bias);

    return 0;
}